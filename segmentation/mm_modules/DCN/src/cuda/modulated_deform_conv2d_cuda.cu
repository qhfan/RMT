#include "hip/hip_runtime.h"
#include <vector>
#include "cuda/modulated_deform_2d_im2col_cuda.cuh"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include <THC/THC.h>
// #include <THC/THCAtomics.cuh>
// #include <THC/THCDeviceUtils.cuh>

// extern THCState *state;

// author: Charles Shang
// https://github.com/torch/cunn/blob/master/lib/THCUNN/generic/SpatialConvolutionMM.cu


at::Tensor
modulated_deform_conv2d_cuda_forward(const at::Tensor &input,
                                     const at::Tensor &weight,
                                     const at::Tensor &bias,
                                     const at::Tensor &offset,
                                     const at::Tensor &mask,
                                     const int kernel_h,
                                     const int kernel_w,
                                     const int stride_h,
                                     const int stride_w,
                                     const int pad_h,
                                     const int pad_w,
                                     const int dilation_h,
                                     const int dilation_w,
                                     const int group,
                                     const int deformable_group,
                                     const int im2col_step)
{
    // THCAssertSameGPU(THCudaTensor_checkGPU(state, 5, input, weight, bias, offset, mask));

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(weight.is_contiguous(), "weight tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h_ = weight.size(2);
    const int kernel_w_ = weight.size(3);

    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);

    AT_ASSERTM((channels % group == 0) && (channels_out % group == 0), 
        "channels(%d) and channels_out(%d) must divide group(%d)", channels, channels_out, group);

    // printf("Kernels: %d %d %d %d\n", kernel_h_, kernel_w_, kernel_w, kernel_h);
    // printf("Channels: %d %d\n", channels, channels_kernel);
    // printf("Channels: %d %d\n", channels_out, channels_kernel);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
               "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == (channels_kernel * group),
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel * group);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    
    auto output = at::empty({batch * height_out * width_out, channels_out}, input.options());

    // prepare group weight and bias
    auto weight_g = weight.view({group, channels_out/group, channels_kernel, kernel_h, kernel_w});
    auto bias_g = bias.view({group, channels_out/group});

    // define alias for easy use
    const int batch_n = im2col_step_;
    const int per_input_size = channels * height * width;
    const int per_offset_size = offset.size(1) * offset.size(2) * offset.size(3);
    const int per_mask_size = mask.size(1) * mask.size(2) * mask.size(3);
    auto output_n = output.view({batch/im2col_step_, batch_n * height_out * width_out, channels_out});
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto columns = at::empty({channels * kernel_h * kernel_w, batch_n * height_out * width_out}, input.options());
        AT_DISPATCH_FLOATING_TYPES(input.type(), "deform_conv_forward_cuda", ([&] {
            modulated_deformable_2d_im2col_cuda(at::cuda::getCurrentCUDAStream(),
                                             input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                             offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                             mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                                             batch_n, channels, height, width,
                                             height_out, width_out, kernel_h, kernel_w,
                                             pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
                                             deformable_group,
                                             columns.data<scalar_t>());

        }));

        auto columns_g = columns.view({group, channels/group * kernel_h * kernel_w, batch_n * height_out * width_out});
        auto output_g = output_n.select(0, n).view({batch_n * height_out * width_out, group, channels_out/group});
        for (int g = 0; g < group; ++g)
        {
            auto columns_gm = columns_g.select(0, g).t();
            auto weight_gm = weight_g.select(0, g).view({channels_out/group, channels_kernel * kernel_h * kernel_w}).t();
            auto output_m = at::addmm(bias_g.select(0, g), columns_gm, weight_gm);
            output_g.select(1, g) = output_m.view({batch_n * height_out * width_out, channels_out/group});
        }

    }

    output = output.view({batch, height_out, width_out, channels_out}).permute({0, 3, 1, 2}).contiguous();

    return output;
}


std::vector<at::Tensor> modulated_deform_conv2d_cuda_backward(const at::Tensor &input,
                                                              const at::Tensor &weight,
                                                              const at::Tensor &bias,
                                                              const at::Tensor &offset,
                                                              const at::Tensor &mask,
                                                              const at::Tensor &grad_output,
                                                              const int kernel_h,
                                                              const int kernel_w,
                                                              const int stride_h,
                                                              const int stride_w,
                                                              const int pad_h,
                                                              const int pad_w,
                                                              const int dilation_h,
                                                              const int dilation_w,
                                                              const int group,
                                                              const int deformable_group,
                                                              const int im2col_step)
{

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(weight.is_contiguous(), "weight tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h_ = weight.size(2);
    const int kernel_w_ = weight.size(3);

    const int batch_ = grad_output.size(0);
    const int channels_out_ = grad_output.size(1);
    const int height_out_ = grad_output.size(2);
    const int width_out_ = grad_output.size(3);

    const int im2col_step_ = std::min(im2col_step, batch);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);

    AT_ASSERTM((channels % group == 0) && (channels_out % group == 0), 
        "channels(%d) and channels_out(%d) must divide group(%d)", channels, channels_out, group);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
               "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == (channels_kernel * group),
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel * group);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    AT_ASSERTM(batch == batch_,
               "Input shape and grad_out batch wont match: (%d vs %d).", batch, batch_);

    AT_ASSERTM(channels_out == channels_out_,
               "Input shape and grad_out channels_out wont match: (%d vs %d).", channels_out, channels_out_);

    AT_ASSERTM(height_out == height_out_ && width_out == width_out_,
               "Input shape and grad_out shape wont match: (%d x %d vs %d x %d).", height_out, height_out_, width_out, width_out_);

    auto ones = at::ones({batch * height_out * width_out}, input.options());
    auto columns = at::empty({channels * kernel_h * kernel_w, batch * 1 * height_out * width_out}, input.options());

    auto grad_input = at::zeros_like(input);
    auto grad_weight = at::zeros_like(weight);
    auto grad_bias = at::zeros_like(bias);
    auto grad_offset = at::zeros_like(offset);
    auto grad_mask = at::zeros_like(mask);

    // prepare group weight and bias
    auto weight_g = weight.view({group, channels_out/group, channels_kernel, kernel_h, kernel_w});
    auto grad_weight_g = grad_weight.view({group, channels_out/group, channels_kernel, kernel_h, kernel_w});
    auto grad_bias_g = grad_bias.view({group, channels_out/group});

    const int batch_n = im2col_step_;
    const int per_input_size = channels * height * width;
    const int per_offset_size = offset.size(1) * offset.size(2) * offset.size(3);
    const int per_mask_size = mask.size(1) * mask.size(2) * mask.size(3);
    auto grad_output_n = grad_output.view({batch/im2col_step_, batch_n, channels_out, height_out, width_out});
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto grad_output_g = grad_output_n.select(0, n).view({batch_n, group, channels_out/group, height_out, width_out});
        auto ones = at::ones({batch_n * height_out * width_out}, input.options());
        auto columns = at::empty({channels * kernel_h * kernel_w, batch_n * 1 * height_out * width_out}, input.options());
        auto columns_g = columns.view({group, channels/group * kernel_h * kernel_w, batch_n * height_out * width_out});
        for (int g = 0; g < group; ++g)
        {
            auto grad_output_gm = grad_output_g.select(1, g).permute({1, 0, 2, 3}).contiguous().view({channels_out/group, batch_n * height_out * width_out});
            auto weight_gm = weight_g.select(0, g).view({channels_out/group, channels_kernel * kernel_h * kernel_w}).t();
            columns_g.select(0, g) = at::mm(weight_gm, grad_output_gm);
        }

        AT_DISPATCH_FLOATING_TYPES(input.type(), "deform_conv_backward_cuda", ([&] {
            modulated_deformable_2d_col2im_coord_cuda(at::cuda::getCurrentCUDAStream(),
                                                   columns.data<scalar_t>(),
                                                   input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                                   offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                                   mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                                                   batch_n, channels, height, width,
                                                   height_out, width_out, kernel_h, kernel_w,
                                                   pad_h, pad_w, stride_h, stride_w,
                                                   dilation_h, dilation_w, deformable_group,
                                                   grad_offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                                   grad_mask.data<scalar_t>() + n * im2col_step_ * per_mask_size);
            // gradient w.r.t. input data
            modulated_deformable_2d_col2im_cuda(at::cuda::getCurrentCUDAStream(),
                                             columns.data<scalar_t>(),
                                             offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                             mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                                             batch_n, channels, height, width,
                                             height_out, width_out, kernel_h, kernel_w,
                                             pad_h, pad_w, stride_h, stride_w,
                                             dilation_h, dilation_w, deformable_group,
                                             grad_input.data<scalar_t>() + n * im2col_step_ * per_input_size);

            // gradient w.r.t. weight, dWeight should accumulate across the batch and group
            modulated_deformable_2d_im2col_cuda(at::cuda::getCurrentCUDAStream(),
                                             input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                             offset.data<scalar_t>() + n * im2col_step_ * per_offset_size,
                                             mask.data<scalar_t>() + n * im2col_step_ * per_mask_size,
                                             batch_n, channels, height, width,
                                             height_out, width_out, kernel_h, kernel_w,
                                             pad_h, pad_w, stride_h, stride_w,
                                             dilation_h, dilation_w, deformable_group,
                                             columns.data<scalar_t>());

        }));

        // auto grad_output_m = grad_output.permute({1, 0, 2, 3}).contiguous().view({channels_out, batch * height_out * width_out});
        // grad_weight = at::mm(grad_output_m, columns.t()).view_as(weight);
        // grad_bias = at::mv(grad_output_m, ones);
        // auto grad_output_g = grad_output.view({batch, group, channels_out/group, height_out, width_out});
        // auto columns_g = columns.view({group, channels/group * kernel_h * kernel_w, batch * height_out * width_out});
        for (int g = 0; g < group; ++g)
        {
            auto grad_output_gm = grad_output_g.select(1, g).permute({1, 0, 2, 3}).contiguous().view({channels_out/group, batch_n * height_out * width_out});
            auto columns_gm = columns_g.select(0, g).t();
            auto grad_weight_gm = grad_weight_g.select(0, g).view({channels_out/group, channels_kernel * kernel_h * kernel_w});
            auto grad_bias_gm = grad_bias_g.select(0, g);
            grad_weight_g.select(0, g) = at::addmm(grad_weight_gm, grad_output_gm, columns_gm).view_as(grad_weight_g.select(0, g));
            grad_bias_g.select(0, g) = at::addmv(grad_bias_gm, grad_output_gm, ones);
        }

    }

    return {
        grad_input, grad_offset, grad_mask, grad_weight, grad_bias
    };
}
